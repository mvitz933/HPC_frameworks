#include "multiple_layers.hpp"



int main(){
	const int shape_length = 4;
	int shape[shape_length] = { 8, 6, 4, 1 };

	// Initialize weights on CPU/RAM
	int nr_weights = 0;

	for (int shape_index = 0; shape_index < shape_length - 1; shape_index++)
	{
		nr_weights += shape[shape_index] * shape[shape_index + 1];
	}

	float *host_weights = new float [nr_weights] {1.62f, -0.61f, -0.53f, -1.07f, 0.87f, -2.30f, 1.74f, -0.76f, 0.32f, -0.25f, 1.46f, -2.06f, -0.32f, -0.38f, 1.13f, -1.10f, -0.17f, -0.88f, 0.04f, 0.58f, -1.10f, 1.14f, 0.90f, 0.50f, 0.90f, -0.68f, -0.12f, -0.94f, -0.27f, 0.53f, -0.69f, -0.40f, -0.69f, -0.85f, -0.67f, -0.01f, -1.12f, 0.23f, 1.66f, 0.74f, -0.19f, -0.89f, -0.75f, 1.69f, 0.05f, -0.64f, 0.19f, 2.10f, 0.12f, 0.62f, 0.30f, -0.35f, -1.14f, -0.35f, -0.21f, 0.59f, 0.84f, 0.93f, 0.29f, 0.89f, -0.75f, 1.25f, 0.51f, -0.30f, 0.49f, -0.08f, 1.13f, 1.52f, 2.19f, -1.40f, -1.44f, -0.50f, 0.16f, 0.88f, 0.32f, -2.02f};

	// Initialize biases on CPU/RAM
	int nr_neurons = 0;
	int nr_biases = 0;

	for (int shape_index = 0; shape_index < shape_length; shape_index++)
	{
		nr_neurons += shape[shape_index];
	}

	nr_biases = nr_neurons - shape[0];
	float *host_biases = new float [nr_biases] {-0.31f, 0.83f, 0.23f, 0.76f, -0.22f, -0.20f, 0.19f, 0.41f, 0.20f, 0.12f, -0.67f};
	
	// The first 8 values are our inputs rest of the array can be initialized with 0.0 
	float *host_activations = new float [nr_neurons] {0.38f, 0.12f, 1.13f, 1.20f, 0.19f, -0.38f, -0.64f, 0.42f};
	
	// Initialize z Matrix
	float *host_z = new float [nr_biases] {0.0f};


	// Calculate the amount of memory needed so we can provide this information to cuda malloc
	const size_t bytes_biases = nr_biases * sizeof(float);
	const size_t bytes_z = nr_biases * sizeof(float);
	const size_t bytes_weights = nr_weights * sizeof(float);
	const size_t bytes_activations = nr_neurons * sizeof(float);
	const size_t bytes_shape = sizeof(int) * shape_length;


	// Allocate GPU device memory
	float *d_biases, *d_weights, *d_activations, *d_z;
	int *d_shape;
	hipMalloc(&d_biases, bytes_biases);
	hipMalloc(&d_weights, bytes_weights);
	hipMalloc(&d_activations, bytes_activations);
	hipMalloc(&d_z, bytes_z);
	hipMalloc(&d_shape, bytes_shape);	

	// Copy data from CPU Memory to GPU Memory
	hipMemcpy(d_biases, host_biases, bytes_biases, hipMemcpyHostToDevice);
	hipMemcpy(d_weights, host_weights, bytes_weights, hipMemcpyHostToDevice);
	hipMemcpy(d_activations, host_activations, bytes_activations, hipMemcpyHostToDevice);
	hipMemcpy(d_z, host_z, bytes_z, hipMemcpyHostToDevice);
	hipMemcpy(d_shape, shape, bytes_shape, hipMemcpyHostToDevice);

	// Call cuda kernel
	int nr_threads = *std::max_element(shape, shape + shape_length);
	linear_layer_and_activation << <1, nr_threads >> > (d_weights, d_biases, d_z, d_activations, d_shape, shape_length);

	// After we caclulated the activations and z values we need to copy the data from GPU Memory back to the CPU Memory
	hipMemcpy(host_activations, d_activations, bytes_activations, hipMemcpyDeviceToHost);
	hipMemcpy(host_z, d_z, bytes_z, hipMemcpyDeviceToHost);

	// Free our memory
	hipFree(d_biases);
	hipFree(d_weights);
	hipFree(d_activations);
	hipFree(d_z);
	hipFree(d_shape);

	int z_offset = 0;
	for (int shape_index = 1; shape_index < shape_length; shape_index++)
	{
		std::cout << "Z Values " << shape_index << ". hidden layer" << std::endl;
		for (int neuron_nr = 0; neuron_nr < shape[shape_index]; neuron_nr++)
		{
			std::cout << host_z[neuron_nr + z_offset] << std::endl;
		}
		z_offset += shape[shape_index];
	}

	int activations_offset = shape[0]; // Skip input values	
	for (int shape_index = 1; shape_index < shape_length; shape_index++)
	{
		std::cout << "Activations " << shape_index << ". hidden layer" << std::endl;

		for (int neuron_nr = 0; neuron_nr < shape[shape_index]; neuron_nr++)
		{
			std::cout << host_activations[neuron_nr + activations_offset] << std::endl;
		}
		activations_offset += shape[shape_index];
	}

	getchar();


	return 0;
}