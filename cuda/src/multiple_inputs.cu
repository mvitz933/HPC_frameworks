#include "hip/hip_runtime.h"
#include "multiple_inputs.hpp"

__global__ void multiple_inputs(float *weight_matrix, float *biases, float *z_values, 
                                float *activation_values, int* shape, int shape_length){
	int id = threadIdx.x;

	// Define offset for the current layer
	int layer_offset_z = 0;
	int layer_offset_b = 0;
	int layer_offset_weights = 0;
	int layer_offset_activations_input_layer = 0;
	int layer_offset_activations_current_layer = shape[0] * blockDim.y;
	
	for (int shape_index = 0; shape_index < shape_length; shape_index++){
		// Other threads don't execute anything to avoid out of bounds access
		if (id < shape[shape_index + 1]){
			int nr_inputs_to_this_layer = shape[shape_index];
			int layer_size = shape[shape_index + 1];
			
			// w*x
			for (int neuron_nr = 0; neuron_nr < nr_inputs_to_this_layer; neuron_nr++){
				z_values[layer_offset_z + threadIdx.y * layer_size + id] += weight_matrix[layer_offset_weights + (nr_inputs_to_this_layer)* id + neuron_nr] *
					activation_values[layer_offset_activations_input_layer + threadIdx.y * nr_inputs_to_this_layer + neuron_nr];
			}

			// w*x + b
			z_values[layer_offset_z + threadIdx.y * layer_size + id] += biases[layer_offset_b + id];

			// sig(w*x + b)	
			// + shape[shape_index] => write activation values for next layer,instead of overwriting the input values                                                
			activation_values[layer_offset_activations_current_layer + layer_size * threadIdx.y + id] = 1.0 / (1.0 + exp(-z_values[layer_offset_z + layer_size * threadIdx.y + id]));
		}

		// Important to do this outside the Memory Guard 
		layer_offset_weights += shape[shape_index] * shape[shape_index + 1];
		layer_offset_b += shape[shape_index + 1];
		layer_offset_z += shape[shape_index + 1] * blockDim.y;
		layer_offset_activations_input_layer = layer_offset_activations_current_layer;
		layer_offset_activations_current_layer += shape[shape_index + 1] * blockDim.y;

		// Call syncthreads so we know that all threads have finished working on the current layer before we take care of the next layer
		// Try removing this and guess what will happen.
		__syncthreads(); 
	}
}